#include "hip/hip_runtime.h"
// #include <iostream>
// #include <sstream>
// #include <cmath>
// #include <ctime>
// #include <numeric>
// #include <limits>
// using namespace std;

// char* getCurrentTimeString() {
// 	std::time_t currTime = std::time(nullptr);
// 	return std::asctime(std::localtime(&currTime));
// }

// #include <array>
// using ResultT = float;
// constexpr ResultT f(int i, int j)
// {
//     return ((float)i / j);
// }

// cuda.cu
#include "cudawrapper.hpp"
#include <iostream>
using namespace std;

// calculates val * [vector elems] and returns if close enough to needle
template<typename T>
__global__ static void compareToZeta5(T *out, const T val, const T needle, const T *coeffArray, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    T expr;
	//printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);

    // Handling arbitrary vector size
    if (tid < n){
		expr = (val * coeffArray[tid]) - needle;
		if (expr < .0000001 && expr > -.0000001) {
			printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
			out[tid] = coeffArray[tid]; //TODO: MAKE THIS ATOMIC AND DYNAMIC instead of only populating "matching" coeffs in array [0, 0, HIT, 0, 0, 0, HIT, etc.]
		}
    }
}

bool InitCUDA(bool b) {
    /* CUDA Initialization */
}

// will calculate ax^5 + bx^4 + cx^3 + dx^2 + ex + f - zeta(5) and return x if
// the value is within a "close enough" bound to zero
// cons = x
// cubicSum = cx^3 + dx^2 + ex + f (pre-calculated)
// coeffArray = array of all possible a/b/c/d/e/f coeff values
// quartLastIndex = last index to loop through for b values
// quintLastIndex = last index to loop through for a values
template<typename T>
std::vector<T>* testForZeta5OnGPU(T cons, T cubicSum, const T *coeffArray, int quartLastIndex, int quintLastIndex)
{
	const T z5 = 1.036927755143369926331365486457034168L; //riemann_zetal((long double)5);
	T currentQuart;
	T quarticSum;
	T consFifth = pow(cons, (T)5);

	T *d_coeffArray, *d_out;
	T *out = new T[quintLastIndex];

	// Allocate device memory 
    hipMalloc((void**)&d_coeffArray, sizeof(T) * quintLastIndex);
    hipMalloc((void**)&d_out, sizeof(T) * quintLastIndex);		//TODO: Can probably make this 10% as large as quintLastIndex??

	cout << "dog\n";

	// Transfer data from host to device memory
    hipMemcpy(d_coeffArray, coeffArray, sizeof(T) * quintLastIndex, hipMemcpyHostToDevice);

	cout << "cat\n";

	int block_size = 1024;
    int grid_size = ((quintLastIndex + block_size) / block_size);
	cout << grid_size << "\n";

	// loop through quarts
	for (int i = 6; i <= 7; i++) { // TODO: Set limit back to quartIndex!
		currentQuart = coeffArray[i];

		quarticSum = cubicSum + currentQuart * pow(cons, (T)4);

		// now we want gpu to calculate ax^5 + quarticSum - zeta(5) and return if close enough to zero
		// or ax^5 + quarticSum == zeta(5) => zeta(5) - quarticSum is close enough to ax^5
		// in other words, figure out what blah - zeta(5) and cons^5 and send those to GPU along with coeff array
		
		// Executing kernel
		compareToZeta5<<<grid_size,block_size>>>(d_out, consFifth, (z5 - quarticSum), coeffArray, quintLastIndex);

		// // Transfer data back to host memory
		hipMemcpy(out, d_out, sizeof(T) * quintLastIndex, hipMemcpyDeviceToHost);
		cout << "ferret\n";
		// //hipDeviceSynchronize();

		for (int j = 0; j < quintLastIndex; j++) {
			if (out[j] < .0000001 && out[j] > .0000001) {
				cout << "i=" << i << ", out[" << j << "] = " << out[j] << "\n";
			}
		}

		//cout << i << "\n";
	}

    
	// cout << "bird\n";

    // Deallocate device memory
    hipFree(d_coeffArray);
    hipFree(d_out);

	cout << "lizard\n";

	delete out;
}

template std::vector<int>* testForZeta5OnGPU<int>(int cons, int cubicSum, const int *coeffArray, int quartLastIndex, int quintLastIndex);
template std::vector<float>* testForZeta5OnGPU<float>(float cons, float cubicSum, const float *coeffArray, int quartLastIndex, int quintLastIndex);
template std::vector<double>* testForZeta5OnGPU<double>(double cons, double cubicSum, const double *coeffArray, int quartLastIndex, int quintLastIndex);